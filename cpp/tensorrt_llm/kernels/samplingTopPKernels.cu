#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{
__global__ void topPInitialize(
    int* topPIdValBuf, int* topPOffsetBuf, int* beginTopPOffsetBuf, int const batchSize, int const vocabSize)
{
    auto const tid = static_cast<int32_t>(threadIdx.x);
    auto const bid = static_cast<int32_t>(blockIdx.x);

    if (bid == 0)
    {
        for (auto i = tid; i < batchSize + 1; i += static_cast<int32_t>(blockDim.x))
        {
            // Inclusive sum of offsets to vocab rows
            topPOffsetBuf[i] = i * vocabSize;
            beginTopPOffsetBuf[i] = topPOffsetBuf[i];
        }
    }

    auto index = tid + bid * static_cast<int32_t>(blockDim.x);

    while (index < batchSize * vocabSize)
    {
        // Set value at {bi, vi} position to vi
        topPIdValBuf[index] = index % vocabSize;
        index += static_cast<int32_t>(blockDim.x * gridDim.x);
    }
}

void invokeTopPInitialize(int* topPIdValBuf, int* topPOffsetBuf, int* beginTopPOffsetBuf, size_t const batchSize,
    int const vocabSize, hipStream_t stream)
{
    // vocabSize: the column number of logits_buffer for top_p sampling
    // TODO(nkorobov): launch based on available resources
    topPInitialize<<<32, 512, 0, stream>>>(topPIdValBuf, topPOffsetBuf, beginTopPOffsetBuf, batchSize, vocabSize);
}

template <typename T, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void topPBeamTopKKernel(T const* logProbs, // prob.
    int* topKTmpIdBuf, T* topKTmpValBuf, FinishedState const* finishedInput, int const vocabSize, int* offsetBuf,
    int* beginOffsetBuf, float const topP, float const* topPs, bool const* skipDecode, int const* batchSlots)
{
    /**
     * Kernel performs top 1 search and saves the token with largest probability if it exceeds probability threshold
     */
    int constexpr MAX_K = 1;
    auto const threadId = static_cast<int32_t>(threadIdx.x);
    auto const batchId = static_cast<int32_t>(blockIdx.x);
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchId] : batchId;

    // Skip decoding kernel if configured
    if ((skipDecode != nullptr && skipDecode[batchSlot])
        || (finishedInput != nullptr && finishedInput[batchSlot].isSkipDecoding()))
    {
        // Required to skip radix sort
        beginOffsetBuf[batchId] += vocabSize;
        return;
    }

    float pThreshold = (topPs != nullptr) ? topPs[batchSlot] : topP;

    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    TopK<T, MAX_K> partial;

    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (int i = 0; i < MAX_K; ++i)
    {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

#pragma unroll
    for (int elemId = threadId; elemId < vocabSize; elemId += THREADBLOCK_SIZE)
    {
        int index = elemId + batchId * vocabSize;
        partial.insert(logProbs[index], elemId);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (threadId == 0)
    {
        beginOffsetBuf[batchId] = offsetBuf[batchId];
        T sumProb = (T) (0.0f);

#pragma unroll
        for (int i = 0; i < MAX_K; i++)
        {
            sumProb += total.u[i];
        }

        if ((float) sumProb >= pThreshold)
        {
            beginOffsetBuf[batchId] += vocabSize;
            int index = batchId * vocabSize;

#pragma unroll
            for (int i = 0; i < MAX_K; ++i)
            {
                topKTmpIdBuf[index + i] = total.p[i];
                topKTmpValBuf[index + i] = total.u[i];
            }
        }
    }
}

struct BlockPrefixCallbackOp
{
    // Running prefix
    float running_total;

    // Constructor
    __device__ BlockPrefixCallbackOp(float running_total)
        : running_total(running_total)
    {
    }

    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide
    // scan.
    __device__ float operator()(float block_aggregate)
    {
        float old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template <typename T>
__device__ void epilogue(int batchId, int currentStep, int offset, int** ids, int* sortedIdVals, T* sortedLogProbs,
    float* cumLogProbs, float* outputLogProbs, int const* endIds, int* sequenceLengths, FinishedState* finishedOutput,
    int maxBatchSize)
{
    ids[batchId][currentStep] = sortedIdVals[offset];

    if (cumLogProbs != nullptr || outputLogProbs != nullptr)
    {
        float lprob = logf(sortedLogProbs[offset]);
        if (cumLogProbs != nullptr)
        {
            cumLogProbs[batchId] += lprob;
        }
        if (outputLogProbs != nullptr)
        {
            outputLogProbs[sequenceLengths[batchId] * maxBatchSize + batchId] = lprob;
        }
    }
    if (sequenceLengths != nullptr && finishedOutput != nullptr)
    {
        if (ids[batchId][currentStep] == endIds[batchId])
        {
            finishedOutput[batchId].setFinishedEOS();
            // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be outputted
        }
        else
        {
            // We don't need to set output finished state as it is assumed to be in non finished state
            sequenceLengths[batchId] += 1;
        }
    }
}

template <typename T, int blockSize>
__global__ void topPSsampling(T* sortedLogProbs, int* sortedIdVals, int** ids, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    int const* beginOffsetBuf, int const* offsetBuf, int const vocabSize, hiprandState_t* curandstate, float const topP,
    float const* topPs, int const* endIds, int maxBatchSize, bool const* skipDecode, int const* batchSlots)
{
    /**
     * Each block processes one request row sorted in descending order by probabilities.
     * All threads within block compute running sum of probabilities until one of the threads exceeds the randomly
     * chosen probability threshold. Thread that crossed probaility threshold writes the corresponding token to the
     * output.
     */

    __shared__ float randNumS;

    auto const tid = static_cast<int32_t>(threadIdx.x);
    auto const batchId = static_cast<int32_t>(blockIdx.x);
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchId] : batchId;
    // Skip kernel if this sampling method is not chosen
    const FinishedState finishState = finishedInput != nullptr ? finishedInput[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    // Exit early if sequence has finished
    if (finishState.isFinished())
    {
        if (tid == 0)
        {
            if (finishedOutput != nullptr)
            {
                finishedOutput[batchSlot] = finishState;
            }
        }
        return;
    }

    constexpr int WARP_SIZE = 32;
    constexpr int NUM_WARPS = blockSize / WARP_SIZE;
    int const laneId = threadIdx.x % WARP_SIZE;
    int const warpId = threadIdx.x / WARP_SIZE;
    float const probThreshold = (topPs != nullptr) ? topPs[batchSlot] : topP;
    int const currentStep = sequenceLength[batchSlot];

    // With P in (0.0; 1.0] we draw a random number P' in range (0.0; P]
    // We will sum all probs moving from the largest probability to the smallest and
    // will choose the token which probability makes cumulative probability sum to exceed P'
    if (threadIdx.x == 0)
    {
        randNumS = hiprand_uniform(curandstate + blockIdx.x) * probThreshold;
    }

    // if beginOffsetBuf and offsetBuf of sorting have same value,
    // this means that we have find best one in topPBeamTopKKernel
    // So, we can skip this sampling.
    if (beginOffsetBuf[batchId] == offsetBuf[batchId])
    {
        if (tid == 0)
        {
            int offset = batchId * vocabSize;
            epilogue(batchSlot, currentStep, offset, ids, sortedIdVals, sortedLogProbs, cumLogProbs, outputLogProbs,
                endIds, sequenceLength, finishedOutput, maxBatchSize);
        }
        return;
    }

    typedef hipcub::BlockScan<float, blockSize> BlockScan;
    __shared__ typename BlockScan::TempStorage tempStorage;
    __shared__ uint32_t selectedShared[NUM_WARPS];
    // Initialize running total
    BlockPrefixCallbackOp prefixOp(0);

    if (laneId == 0)
    {
        selectedShared[warpId] = 0;
    }

    __syncthreads();

    int offset = batchId * vocabSize;
    ids[batchSlot][currentStep] = sortedIdVals[offset];
    int end = ((vocabSize + blockSize - 1) / blockSize) * blockSize;
    int selectedTokenId = 0;
    // Cumulative sum
    float threadOffset = 0;
    int count = 0;
    for (int vi = tid; vi < end; vi += blockSize)
    {
        float threadProb = (vi < vocabSize) ? (float) sortedLogProbs[offset + vi] : 0.f;
        BlockScan(tempStorage).InclusiveSum(threadProb, threadOffset, prefixOp);
        count = __syncthreads_count(randNumS <= threadOffset);
        selectedTokenId = vi;
        if (count != 0)
        {
            break;
        }
    }

    // select first thread exceeded the prob threshold or the last thread in case of P=1.0f
    if (threadIdx.x == min(blockDim.x - count, blockDim.x - 1))
    {
        epilogue(batchSlot, currentStep, offset + selectedTokenId, ids, sortedIdVals, sortedLogProbs, cumLogProbs,
            outputLogProbs, endIds, sequenceLength, finishedOutput, maxBatchSize);
    }
}

template <typename T>
std::vector<size_t> getTopPWorkspaceSizes(int32_t batchSize, int32_t vocabSize)
{
    auto const sortedLogProbBufSize = sizeof(T) * batchSize * vocabSize;      // type T
    auto const sortedIdValsBufSize = sizeof(int32_t) * batchSize * vocabSize; // type int

    size_t cubTempStorageSize;
    tensorrt_llm::common::check_cuda_error(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr,
        cubTempStorageSize, static_cast<T*>(nullptr), static_cast<T*>(nullptr), static_cast<int32_t*>(nullptr),
        static_cast<int32_t*>(nullptr), static_cast<int32_t>(vocabSize * batchSize), batchSize,
        static_cast<int32_t*>(nullptr), static_cast<int32_t*>(nullptr),
        0,             // begin_bit
        sizeof(T) * 8, // end_bit = sizeof(KeyT) * 8
        0));           // hipStream_t

    return {cubTempStorageSize, sortedLogProbBufSize, sortedIdValsBufSize};
}

template std::vector<size_t> getTopPWorkspaceSizes<float>(int32_t batchSize, int32_t vocabSize);
template std::vector<size_t> getTopPWorkspaceSizes<half>(int32_t batchSize, int32_t vocabSize);

template <typename T>
size_t getTopPWorkspaceSize(int32_t batchSize, int32_t vocabSizePadded)
{
    auto const workspaceSizes = getTopPWorkspaceSizes<T>(batchSize, vocabSizePadded);
    return tensorrt_llm::common::calcAlignedSize(workspaceSizes, 256);
}

template size_t getTopPWorkspaceSize<float>(int32_t batchSize, int32_t vocabSizePadded);
template size_t getTopPWorkspaceSize<half>(int32_t batchSize, int32_t vocabSizePadded);

template <typename T>
void invokeBatchTopPSampling(void* workspace, int** outputIds, int* sequenceLength, FinishedState const* finishedInput,
    FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs, T const* logProbs, int32_t const* idVals,
    int* offsetBuf, int* beginOffsetBuf, hiprandState_t* curandstate, int const batchSize, int maxBatchSize,
    size_t const vocabSize, int const* endIds, float const maxTopP, float const* topPs, hipStream_t stream,
    bool const* skipDecode, int const* batchSlots)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto const workspaceSizes = getTopPWorkspaceSizes<T>(batchSize, vocabSize);

    std::vector<void*> alignedPointers;
    calcAlignedPointers(alignedPointers, workspace, workspaceSizes);

    auto cubTempStorage = static_cast<void*>(alignedPointers[0]);
    auto sortedLogProbs = static_cast<T*>(alignedPointers[1]);
    auto sortedIdVals = static_cast<int32_t*>(alignedPointers[2]);

    int constexpr BLOCK_SIZE = 256;
    // Performs Top K=1 search.
    // If the most probable token exceeds P, we skip sorting by setting beginOffsetBuf[bi] = offsetBuf[bi]
    topPBeamTopKKernel<T, BLOCK_SIZE><<<batchSize, BLOCK_SIZE, 0, stream>>>(logProbs, sortedIdVals, sortedLogProbs,
        finishedInput, vocabSize, offsetBuf, beginOffsetBuf, maxTopP, topPs, skipDecode, batchSlots);
    sync_check_cuda_error();

    // Sort tokens by probability in descending order
    auto cubWorkspaceSize = workspaceSizes[0];
    check_cuda_error(
        hipcub::DeviceSegmentedRadixSort::SortPairsDescending(cubTempStorage, cubWorkspaceSize, logProbs, sortedLogProbs,
            idVals, sortedIdVals, static_cast<int32_t>(vocabSize * batchSize), batchSize, beginOffsetBuf, offsetBuf + 1,
            0,                                   // begin_bit
            static_cast<int32_t>(sizeof(T) * 8), // end_bit = sizeof(KeyT) * 8
            stream));                            // hipStream_t

    int constexpr SAMPLING_BLOCK_SIZE = 256;
    dim3 grid(batchSize);
    // Sample with Top P given sorted tokens
    topPSsampling<T, SAMPLING_BLOCK_SIZE><<<grid, SAMPLING_BLOCK_SIZE, 0, stream>>>(sortedLogProbs, sortedIdVals,
        outputIds, sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, beginOffsetBuf,
        offsetBuf + 1, vocabSize, curandstate, maxTopP, topPs, endIds, maxBatchSize, skipDecode, batchSlots);
    sync_check_cuda_error();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template void invokeBatchTopPSampling(void* workspace, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    float const* logProbs, int32_t const* idVals, int* offsetBuf, int* beginOffsetBuf, hiprandState_t* curandstate,
    int const batchSize, int maxBatchSize, size_t const vocabSizePadded, int const* endIds, float const maxTopP,
    float const* topPs, hipStream_t stream, bool const* skipDecode, int const* batchSlots);

template void invokeBatchTopPSampling(void* workspace, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    half const* logProbs, int32_t const* idVals, int* offsetBuf, int* beginOffsetBuf, hiprandState_t* curandstate,
    int const batchSize, int maxBatchSize, size_t const vocabSizePadded, int const* endIds, float const maxTopP,
    float const* topPs, hipStream_t stream, bool const* skipDecode, int const* batchSlots);

template <typename T>
void invokeTopPSampling(void* workspace, int** outputIds, int* sequenceLength, FinishedState const* finishedInput,
    FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs, T const* logProbs, int32_t const* idVals,
    int* offsetBuf, int* beginOffsetBuf, hiprandState_t* curandstate, int const batchSize, int maxBatchSize,
    size_t const vocabSizePadded, int const* endIds, float const topP, hipStream_t stream, bool const* skipDecode,
    int const* batchSlots)
{
    invokeBatchTopPSampling(workspace, outputIds, sequenceLength, finishedInput, finishedOutput, cumLogProbs,
        outputLogProbs, logProbs, idVals, offsetBuf, beginOffsetBuf, curandstate, batchSize, maxBatchSize,
        vocabSizePadded, endIds, topP, nullptr, stream, skipDecode, batchSlots);
}

template void invokeTopPSampling(void* workspace, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    float const* logProbs, int32_t const* idVals, int* offsetBuf, int* beginOffsetBuf, hiprandState_t* curandstate,
    int const batchSize, int maxBatchSize, size_t const vocabSizePadded, int const* endIds, float const topP,
    hipStream_t stream, bool const* skipDecode, int const* batchSlots);

template void invokeTopPSampling(void* workspace, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    half const* logProbs, int32_t const* idVals, int* offsetBuf, int* beginOffsetBuf, hiprandState_t* curandstate,
    int const batchSize, int maxBatchSize, size_t const vocabSizePadded, int const* endIds, float const topP,
    hipStream_t stream, bool const* skipDecode, int const* batchSlots);

__global__ void computeToppDecay(float* runtimeTopP, float const* runtimeInitialTopP, int const** outputIds,
    float const* topPDecay, float const* topPMin, int32_t const* topPResetIds, int const* sequenceLengths,
    int const* batchSlots)
{
    int const idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[idx] : idx;
    auto const currentStep{sequenceLengths[batchSlot]};
    if (outputIds[batchSlot][currentStep] == topPResetIds[batchSlot])
    {
        runtimeTopP[batchSlot] = runtimeInitialTopP[batchSlot];
    }
    else
    {
        runtimeTopP[batchSlot] = max(runtimeTopP[batchSlot] * topPDecay[batchSlot], topPMin[batchSlot]);
    }
}

void invokeComputeToppDecay(float* runtimeTopP, float const* runtimeInitialTopP, int const** outputIds,
    float const* topPDecay, float const* topPMin, int32_t const* topPResetIds, int const* sequenceLengths,
    int const* batchSlots, int const localBatchSize, hipStream_t stream)
{
    dim3 block(min(localBatchSize, 512));
    dim3 grid((localBatchSize + block.x - 1) / block.x);
    computeToppDecay<<<grid, block, 0, stream>>>(
        runtimeTopP, runtimeInitialTopP, outputIds, topPDecay, topPMin, topPResetIds, sequenceLengths, batchSlots);
}

} // namespace kernels
} // namespace tensorrt_llm
