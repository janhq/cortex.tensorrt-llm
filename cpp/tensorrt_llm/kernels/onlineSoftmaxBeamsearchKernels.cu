/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/onlineSoftmaxBeamsearchKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T, int MAX_K>
void topK_softMax_kernelLauncher(T const* log_probs, T const* bias, void* temp_storage, int const temp_storage_size,
    BeamHypotheses& beam_hyps, hipStream_t stream);

#define CASE_K(MAX_K)                                                                                                  \
    topK_softMax_kernelLauncher<T, MAX_K>(log_probs, bias, temp_storage, temp_storage_size, beam_hyps, stream);        \
    break;

template <typename T>
void invokeTopkSoftMax(T const* log_probs, T const* bias, void* temp_storage, int const temp_storage_size,
    BeamHypotheses& beam_hyps, hipStream_t stream)
{
    int log_beam_width(0);
    int recursor(beam_hyps.beam_width - 1);
    while (recursor >>= 1)
        ++log_beam_width;

    switch (log_beam_width)
    {
    case 0:
    case 1:        // 0 < beam_width <= 4
        CASE_K(4)
    case 2:        // 4 < beam_width <= 8
        CASE_K(8)
#ifndef FAST_BUILD // For fast build, skip case 3, 4, 5
    case 3:        // 9 < beam_width <= 16
        CASE_K(16)
    case 4:        // 16 < beam_width <= 32
        CASE_K(32)
    case 5:        // 32 < beam_width <= 64
        CASE_K(64)
#endif             // FAST_BUILD
    default:
        throw std::runtime_error(fmtstr("%s:%d Topk kernel of beam search does not support beam_width=%d", __FILE__,
            __LINE__, beam_hyps.beam_width));
    }
}

#undef CASE_K

template void invokeTopkSoftMax<float>(float const* log_probs, float const* bias, void* tmp_storage,
    int const temp_storage_size, BeamHypotheses& beam_hyps, hipStream_t stream);

template void invokeTopkSoftMax<half>(half const* log_probs, half const* bias, void* tmp_storage,
    int const temp_storage_size, BeamHypotheses& beam_hyps, hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
