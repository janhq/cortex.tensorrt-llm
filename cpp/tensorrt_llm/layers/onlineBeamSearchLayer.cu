#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/beamSearchTopkKernels.h"
#include "tensorrt_llm/layers/fillBuffers.h"
#include "tensorrt_llm/layers/onlineBeamSearchLayer.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

static int const SMALL_TOP_K_SOFTMAX_MAX_VOC_PARTS = 128;
static int const MAX_K = 4;

template <typename T>
__global__ void update_kernel(BeamHypotheses beam_hyps)
{
    int const beam_width{beam_hyps.beam_width};
    int const ite{beam_hyps.ite};
    int const local_batch_size{beam_hyps.local_batch_size};
    int const max_seq_len{beam_hyps.max_seq_len};
    int const vocab_size{beam_hyps.vocab_size};
    int const end_id{beam_hyps.end_ids[blockIdx.x]};
    int* num_beams{beam_hyps.num_beams};
    int* sequence_lengths{beam_hyps.sequence_lengths_src};
    int** output_ids_ptr{beam_hyps.output_ids_tgt_ptr};
    int** parent_ids_ptr{beam_hyps.parent_ids_tgt_ptr};
    FinishedState* finished{beam_hyps.finished};

    extern __shared__ char s_buf[]; // intermediate result
    int* s_sequence_lengths = reinterpret_cast<int*>(s_buf);

    for (int beam_idx = threadIdx.x; beam_idx < beam_width; beam_idx += blockDim.x)
    {
        auto const batch_beam_idx = blockIdx.x * beam_width + beam_idx;
        s_sequence_lengths[beam_idx] = sequence_lengths[batch_beam_idx];
    }
    __syncthreads();

    for (int beam_idx = threadIdx.x; beam_idx < beam_width; beam_idx += blockDim.x)
    {
        auto const batch_beam_idx = blockIdx.x * beam_width + beam_idx;
        int const current_step{s_sequence_lengths[beam_idx]};

        // Increase the seq_len even if the request has finished.
        // On the following iteration we check if the sequence has finished before
        auto const finish_state = finished[batch_beam_idx];
        if (!finish_state.isFinished())
        {
            s_sequence_lengths[beam_idx]++;
        }

        int new_word_id{output_ids_ptr[blockIdx.x][beam_idx * max_seq_len + current_step]};
        int new_beam_id{(new_word_id / vocab_size) % beam_width};
        new_word_id = new_word_id % vocab_size;

        sequence_lengths[batch_beam_idx] = s_sequence_lengths[new_beam_id];
        if (new_word_id == end_id)
        {
            finished[batch_beam_idx].setFinishedEOS();
        }
        parent_ids_ptr[blockIdx.x][beam_idx * max_seq_len + current_step] = new_beam_id;
        output_ids_ptr[blockIdx.x][beam_idx * max_seq_len + current_step] = new_word_id;
    }
    if (num_beams != nullptr && num_beams[ite * local_batch_size + blockIdx.x] == beam_width)
    {
        for (int beam_idx = threadIdx.x; beam_idx < beam_width; beam_idx += blockDim.x)
        {
            finished[blockIdx.x * beam_width + beam_idx].setFinished();
        }
    }
}

void invokeUpdate(BeamHypotheses& beam_hyps, hipStream_t stream)
{
    dim3 grid(beam_hyps.local_batch_size);
    dim3 block(min(beam_hyps.beam_width, 1024));
    update_kernel<float><<<grid, block, sizeof(int) * beam_hyps.beam_width, stream>>>(beam_hyps);
}

template <typename T>
void OnlineBeamSearchLayer<T>::setup(size_t batch_size, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    BaseBeamSearchLayer<T>::setupBase(batch_size, setupParams);
    allocateBuffer(batch_size);

    mDiversityRate.resize(batch_size);
    mLengthPenalty.resize(batch_size);
    mEarlyStopping.resize(batch_size);
    FillBuffers const fillBuffers{batch_size, batch_size, mStream};

    fillBuffers(setupParams.beam_search_diversity_rate, 0.0f, mDiversityRate, diversity_rates_buf_, (int*) nullptr);
    fillBuffers(setupParams.length_penalty, 0.0f, mLengthPenalty, length_penalties_buf_, (int*) nullptr);
    fillBuffers(setupParams.early_stopping, 1, mEarlyStopping, early_stoppings_buf_, (int*) nullptr);
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void OnlineBeamSearchLayer<T>::invokeSoftMax(BeamSearchOutputParams& outputs, SoftmaxParams const& params)
{
    TLLM_LOG_TRACE("%s", __PRETTY_FUNCTION__);

    BeamHypotheses beam_hyps;
    if (outputs.beamHypotheses)
    {
        beam_hyps = *outputs.beamHypotheses;
        beam_hyps.end_ids = params.end_ids.template getPtr<int const>();
        beam_hyps.finished
            = reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>());
        beam_hyps.cum_log_probs_src = outputs.cum_log_probs->template getPtr<float>();
        beam_hyps.log_probs_src
            = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;
        beam_hyps.sequence_lengths_src = outputs.sequence_length->template getPtr<int>();
        beam_hyps.output_ids_tgt_ptr = outputs.output_ids_ptr.template getPtr<int*>();
        beam_hyps.parent_ids_tgt_ptr = outputs.parent_ids_ptr.template getPtr<int*>();

        beam_hyps.diversity_rates = diversity_rates_buf_;
        beam_hyps.length_penalties = length_penalties_buf_;
        beam_hyps.early_stoppings = early_stoppings_buf_;

        beam_hyps.batch_size = static_cast<std::int32_t>(outputs.output_ids_ptr.shape[0]);
        beam_hyps.beam_width = static_cast<std::int32_t>(outputs.output_ids_ptr.shape[1]);
        beam_hyps.ite = params.ite;
        beam_hyps.local_batch_size = params.logits.shape[0];
        beam_hyps.max_seq_len = static_cast<std::int32_t>(outputs.output_ids_ptr.shape[2]);
        beam_hyps.vocab_size = vocab_size_padded_;
    }

    T const* logits = params.logits.template getPtr<T>();
    T const* bias = static_cast<T const*>(nullptr);

    invokeTopkSoftMax(logits, bias, topk_softmax_workspace_, topk_softmax_workspace_size_, beam_hyps, mStream);
    sync_check_cuda_error();

    invokeUpdate(beam_hyps, mStream);
    sync_check_cuda_error();
}

template <typename T>
void OnlineBeamSearchLayer<T>::allocateBuffer(size_t batch_size)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    // we need to check 2 * beam_width candidates each time
    // 64 is the max beam width we support now.
    topk_softmax_workspace_size_ = (size_t) (ceil(batch_size * 64 * (64 * 2) / 4.) * 4 * 2
        + ceil(batch_size * (64 * 2) * SMALL_TOP_K_SOFTMAX_MAX_VOC_PARTS * (2 * (MAX_K * 2) + 2) / 4.) * 4);

    topk_softmax_workspace_ = reinterpret_cast<float*>(
        mAllocator->reMalloc(topk_softmax_workspace_, sizeof(float) * topk_softmax_workspace_size_, true));
    diversity_rates_buf_ = mAllocator->reMalloc(diversity_rates_buf_, sizeof(float) * batch_size, false);
    length_penalties_buf_ = mAllocator->reMalloc(length_penalties_buf_, sizeof(float) * batch_size, false);
    early_stoppings_buf_ = mAllocator->reMalloc(early_stoppings_buf_, sizeof(int) * batch_size, false);

    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void OnlineBeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&topk_softmax_workspace_));
        mAllocator->free((void**) (&diversity_rates_buf_));
        mAllocator->free((void**) (&length_penalties_buf_));
        mAllocator->free((void**) (&early_stoppings_buf_));
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
OnlineBeamSearchLayer<T>::OnlineBeamSearchLayer(
    size_t vocab_size, size_t vocab_size_padded, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseBeamSearchLayer<T>(vocab_size, vocab_size_padded, stream, std::move(allocator))
{
}

template <typename T>
OnlineBeamSearchLayer<T>::OnlineBeamSearchLayer(OnlineBeamSearchLayer<T> const& beam_search_layer)
    : BaseBeamSearchLayer<T>(beam_search_layer)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
OnlineBeamSearchLayer<T>::~OnlineBeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template class OnlineBeamSearchLayer<float>;
template class OnlineBeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
