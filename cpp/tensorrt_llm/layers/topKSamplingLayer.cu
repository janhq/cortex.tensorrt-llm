#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topKSamplingLayer.h"
#include "tensorrt_llm/runtime/iTensor.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace layers
{

template <uint32_t TOP_K_MAX>
__global__ void setupTopKRuntimeArgs(int batchSize, uint32_t topK, uint32_t* topKs, int topKsSize, float topP,
    float* topPs, int topPsSize, bool* skipDecode, int const* batchSlots)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int bi = index; bi < batchSize; bi += gridDim.x * blockDim.x)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        uint32_t k = topKsSize > 1 ? topKs[batchSlot] : topK;
        float p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f)
        {
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX.
        topKs[batchSlot] = k;
        // Clip p value if it is out of range. range = [0.0, 1.0].
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k == 0;
    }
}

template <typename T>
void TopKSamplingLayer<T>::allocateBuffer(size_t const batchSize)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    mSamplingWorkspaceSize = getTopKWorkspaceSize<T>(batchSize, 1, TOP_K_MAX, mVocabSizePadded);

    std::array<size_t, 4> deviceBufferSizes;
    deviceBufferSizes[0] = sizeof(uint32_t) * batchSize;
    deviceBufferSizes[1] = sizeof(float) * batchSize;
    deviceBufferSizes[2] = sizeof(bool) * batchSize;
    deviceBufferSizes[3] = std::max(deviceBufferSizes[0], deviceBufferSizes[1]);

    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[0], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[1], false);
    mSkipDecodeDevice = mAllocator->reMalloc(mSkipDecodeDevice, deviceBufferSizes[2], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[3], false);

    mSkipDecodeHost = (bool*) std::realloc(mSkipDecodeHost, sizeof(bool) * batchSize);

    mAllocatedSize = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topKSamplingLayer allocated %lu bytes on GPU", mAllocatedSize);
}

template <typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    mAllocator->free((void**) (&mRuntimeTopKDevice));
    mAllocator->free((void**) (&mRuntimeTopPDevice));
    mAllocator->free((void**) (&mSkipDecodeDevice));
    mAllocator->free((void**) (&mSetupWorkspaceDevice));
    std::free(mSkipDecodeHost);
}

template <typename T>
void TopKSamplingLayer<T>::setup(size_t const batchSize, int32_t const* batchSlots, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);

    uint32_t constexpr defaultTopK = 0;
    auto runtimeTopK = setupParams.runtime_top_k.value_or(std::vector<uint32_t>{defaultTopK});
    auto runtimeTopP = setupParams.runtime_top_p.value_or(std::vector<float>{});

    size_t const runtimeTopKSize = runtimeTopK.size();
    size_t const runtimeTopPSize = runtimeTopP.size();
    mNormalizeLogProbs = setupParams.normalize_log_probs.has_value() && setupParams.normalize_log_probs.value();

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }
    for (auto& topK : runtimeTopK)
    {
        if (topK > TOP_K_MAX)
        {
            TLLM_LOG_WARNING(
                "TopK (%d) is larger than max supported number (%d). Clip to max supported number.", topK, TOP_K_MAX);
            topK = TOP_K_MAX;
        }
    }

    uint32_t const topK = *std::max_element(std::begin(runtimeTopK), std::end(runtimeTopK));
    float const topP = (runtimeTopPSize == 0) ? 0.0f : runtimeTopP.front();

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopK.size() == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<uint32_t*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<uint32_t*>(mSetupWorkspaceDevice), mRuntimeTopKDevice, batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopP.size() == batchSize,
            fmtstr("runtimeTopP.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    {
        dim3 block(std::min((int) batchSize, 256));
        dim3 grid(divUp((int) batchSize, (int) block.x));
        // support topK up to TOP_K_MAX.
        setupTopKRuntimeArgs<TOP_K_MAX><<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice,
            runtimeTopKSize, topP, mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots);
    }

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mMaxBatchSize, mStream);
    std::vector<uint32_t> runtimeTopKs(mMaxBatchSize);
    cudaAutoCpy(runtimeTopKs.data(), mRuntimeTopKDevice, mMaxBatchSize, mStream);
    {
        uint32_t maxTopK = 0;
        for (size_t bi = 0; bi < batchSize; ++bi)
        {
            uint32_t bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            maxTopK = std::max(maxTopK, runtimeTopKs[bid]);
        }
        mRuntimeMaxTopK = std::max(mRuntimeMaxTopK, maxTopK);
    }
}

template <typename T>
void TopKSamplingLayer<T>::forward(DecodingOutputParams& outputs, ForwardParams& inputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto const batchSize = inputs.logits.shape[0];

    auto logits = inputs.logits.template getPtr<T>();
    auto endIds = inputs.end_ids.template getPtr<int const>();
    auto batchSlots = inputs.batch_slots ? inputs.batch_slots->template getPtr<int const>() : nullptr;
    auto curandStatesDevice = inputs.curand_states;
    auto samplingWorkspaceDevice = inputs.sampling_workspace;
    auto const probsComputed = inputs.probs_computed;

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    FinishedState* finishedInput = (inputs.finished)
        ? reinterpret_cast<FinishedState*>(inputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs.finished)
        ? reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;

    float* cumLogProbs = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : nullptr;
    float* outputLogProbs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;
    int* sequenceLength = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeBatchTopKSampling(samplingWorkspaceDevice, logits, static_cast<T const* const*>(nullptr),
        outputs.output_ids_ptr.template getPtr<int*>(), sequenceLength, finishedInput, finishedOutput, cumLogProbs,
        outputLogProbs, curandStatesDevice, static_cast<int32_t>(mRuntimeMaxTopK),
        reinterpret_cast<int32_t*>(mRuntimeTopKDevice), 1.0f, mRuntimeTopPDevice, mVocabSizePadded, endIds, batchSlots,
        mStream, batchSize, mMaxBatchSize, nullptr, 1, mSkipDecodeDevice, mNormalizeLogProbs, probsComputed,
        /* return all Top-K*/ false);
    sync_check_cuda_error();
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(size_t maxBatchSize, size_t vocabSize, size_t vocabSizePadded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseSamplingLayer<T>(maxBatchSize, vocabSize, vocabSizePadded, stream, std::move(allocator), nullptr)
{
    allocateBuffer(mMaxBatchSize);
}

template <typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
