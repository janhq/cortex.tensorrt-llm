#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/layers/baseBeamSearchLayer.h"

#include <algorithm>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

__global__ void update_indir_cache_kernel(int* tgt_indir_cache, int const* src_indir_cache, int const** parent_ids,
    FinishedState const* finished, int const* sequence_lengths, int const* input_lengths, int batch_dim,
    int local_batch_size, int beam_width, int max_attention_window, int sink_token_length, int max_seq_len)
{
    int time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int bb_id = threadIdx.y + blockIdx.y * blockDim.y;   // should be just blockIdx.y?
    int const current_step{sequence_lengths[bb_id] - 1}; // the sequence_lengths is updated, need to minus 1
    int const input_length{input_lengths == nullptr ? 0 : input_lengths[bb_id]};
    int const batch_id = bb_id / beam_width;
    int const beam_id = bb_id % beam_width;
    // Exit when the batch_beam or timestep is out of the bound.
    // Assume that KV Cache is shared and fixed for context part,
    //  so we don't need to update the indices for context part.
    if (bb_id >= beam_width * local_batch_size || time_step >= max_seq_len || time_step < input_length
        || time_step < (max_seq_len - max_attention_window) || finished[bb_id].isFinished())
    {
        return;
    }
    int time_step_circ = time_step;
    if (time_step_circ >= sink_token_length)
    {
        time_step_circ
            = sink_token_length + (time_step - sink_token_length) % (max_attention_window - sink_token_length);
    }

    // for the parent_ids, we will still keep it for all past tokens (i.e. max_seq_len)
    int const src_beam = parent_ids[batch_id][beam_id * max_seq_len + current_step];

    // for the indir tables, we have the cyclic kv cache.
    const uint32_t tgt_offset
        = batch_id * beam_width * max_attention_window + beam_id * max_attention_window + time_step_circ;
    const uint32_t src_offset
        = batch_id * beam_width * max_attention_window + src_beam * max_attention_window + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void update_indir_cache_kernelLauncher(int* tgt_indir_cache, int const* src_indir_cache, int const** parent_ids,
    FinishedState const* finished, int const* sequence_lengths, int const* input_lengths, int batch_dim,
    int local_batch_size, int beam_width, int max_seq_len, int max_attention_window, int sink_token_length,
    hipStream_t stream)
{
    const dim3 block(32);
    // Update indirections steps [input_length[bb_id], sequence_lengths[bb_id]], included
    const dim3 grid((max_seq_len + block.x - 1) / block.x, local_batch_size * beam_width);
    update_indir_cache_kernel<<<grid, block, 0, stream>>>(tgt_indir_cache, src_indir_cache, parent_ids, finished,
        sequence_lengths, input_lengths, batch_dim, local_batch_size, beam_width, max_attention_window,
        sink_token_length, max_seq_len);
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(
    size_t vocab_size, size_t vocab_size_padded, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(stream, std::move(allocator), nullptr)
    , vocab_size_(vocab_size)
    , vocab_size_padded_(vocab_size_padded)
{
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(BaseBeamSearchLayer<T> const& beam_search_layer)
    : BaseLayer(beam_search_layer)
    , vocab_size_(beam_search_layer.vocab_size_)
    , vocab_size_padded_(beam_search_layer.vocab_size_padded_)
    , topk_softmax_workspace_size_(beam_search_layer.topk_softmax_workspace_size_)
{
}

template <typename T>
BaseBeamSearchLayer<T>::~BaseBeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template <typename T>
void BaseBeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::allocateBuffer(size_t batch_size)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::setupBase(size_t batch_size, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    allocateBuffer(batch_size);
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::forward(BeamSearchOutputParams& outputs, ForwardParams const& params)
{
    TLLM_LOG_TRACE("%s", __PRETTY_FUNCTION__);
    Tensor& output_ids_ptr = outputs.output_ids_ptr;

    auto const batch_size = static_cast<std::int32_t>(output_ids_ptr.shape[0]);
    auto const beam_width = static_cast<std::int32_t>(output_ids_ptr.shape[1]);
    auto const max_seq_len = static_cast<std::int32_t>(output_ids_ptr.shape[2]);

    TLLM_CHECK_WITH_INFO(params.ite == 0, "Pipeline Parallelism is not supported yet !");

    int const ite = params.ite;
    auto* const input_lengths = params.input_lengths ? params.input_lengths->template getPtr<int const>() : nullptr;
    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;
    Tensor const& logits = params.logits;
    auto const local_batch_size = logits.shape[0];

    invokeSoftMax(outputs, params);
    sync_check_cuda_error();

    if (beam_width > 1)
    {
        update_indir_cache_kernelLauncher(outputs.tgt_cache_indirection.template getPtr<int>(),
            params.src_cache_indirection.template getPtr<int const>(),
            outputs.parent_ids_ptr.template getPtr<int const*>(),
            reinterpret_cast<FinishedState const*>(
                outputs.finished->template getPtr<const FinishedState::UnderlyingType>()),
            sequence_length, input_lengths, batch_size, local_batch_size, beam_width, max_seq_len,
            params.max_attention_window, params.sink_token_length, mStream);
        sync_check_cuda_error();
    }
}

template class BaseBeamSearchLayer<float>;
template class BaseBeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
