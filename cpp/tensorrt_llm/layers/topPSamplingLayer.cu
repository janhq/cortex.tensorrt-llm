#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingAirTopPKernels.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topPSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

static __global__ void setTopPRuntimeArgs(int batchSize, uint32_t topK, uint32_t* topKs, int topKsSize, float topP,
    float* topPs, int topPsSize, bool* skipDecode, int const* batchSlots, float* initialTopPBuf)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
              and top_k to top_ks.
     */

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int bi = index; bi < batchSize; bi += gridDim.x * blockDim.x)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        std::uint32_t k = topKsSize > 1 ? topKs[batchSlot] : topK;
        float p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        topKs[batchSlot] = k;
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k > 0;

        initialTopPBuf[batchSlot] = topPs[batchSlot];
    }
}

template <typename T>
void TopPSamplingLayer<T>::allocateBuffer(size_t batchSize)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    if (mIsDeterministic)
    {
        mSamplingWorkspaceSize = getTopPWorkspaceSize<T>(batchSize, mVocabSizePadded);
    }
    else
    {
        mSamplingWorkspaceSize = getAirTopPWorkspaceSize<T>(batchSize, mVocabSizePadded);
    }

    std::array<size_t, 11> deviceBufferSizes;
    deviceBufferSizes[0] = sizeof(int32_t) * batchSize * mVocabSizePadded;
    deviceBufferSizes[1] = sizeof(int32_t) * (batchSize + 1);
    deviceBufferSizes[2] = sizeof(int32_t) * (batchSize + 1);
    deviceBufferSizes[3] = sizeof(uint32_t) * batchSize;
    deviceBufferSizes[4] = sizeof(float) * batchSize;
    deviceBufferSizes[5] = sizeof(float) * batchSize;
    deviceBufferSizes[6] = sizeof(float) * batchSize;
    deviceBufferSizes[7] = sizeof(float) * batchSize;
    deviceBufferSizes[8] = sizeof(int32_t) * batchSize;
    deviceBufferSizes[9] = sizeof(bool) * batchSize;
    deviceBufferSizes[10] = *std::max_element(&deviceBufferSizes[3], &deviceBufferSizes[9]);

    mTopPIdValsDevice = mAllocator->reMalloc(mTopPIdValsDevice, deviceBufferSizes[0], false);
    mTopPOffsetDevice = mAllocator->reMalloc(mTopPOffsetDevice, deviceBufferSizes[1], false);
    mBeginTopPOffsetDevice = mAllocator->reMalloc(mBeginTopPOffsetDevice, deviceBufferSizes[2], false);
    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[3], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[4], false);
    mInitialTopPDevice = mAllocator->reMalloc(mInitialTopPDevice, deviceBufferSizes[5], false);
    mTopPDecayDevice = mAllocator->reMalloc(mTopPDecayDevice, deviceBufferSizes[6], false);
    mTopPMinDevice = mAllocator->reMalloc(mTopPMinDevice, deviceBufferSizes[7], false);
    mTopPResetIdsDevice = mAllocator->reMalloc(mTopPResetIdsDevice, deviceBufferSizes[8], false);
    mSkipDecodeDevice = mAllocator->reMalloc(mSkipDecodeDevice, deviceBufferSizes[9], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[10], false);

    mSkipDecodeHost = (bool*) std::realloc(mSkipDecodeHost, sizeof(bool) * batchSize);
    std::fill(mSkipDecodeHost, mSkipDecodeHost + batchSize, true);

    mAllocatedSize = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topPSamplingLayer allocated %lu bytes on GPU", mAllocatedSize);
}

template <typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    mAllocator->free((void**) (&mTopPIdValsDevice));
    mAllocator->free((void**) (&mTopPOffsetDevice));
    mAllocator->free((void**) (&mBeginTopPOffsetDevice));
    mAllocator->free((void**) (&mRuntimeTopKDevice));
    mAllocator->free((void**) (&mRuntimeTopPDevice));
    mAllocator->free((void**) (&mInitialTopPDevice));
    mAllocator->free((void**) (&mTopPDecayDevice));
    mAllocator->free((void**) (&mTopPMinDevice));
    mAllocator->free((void**) (&mTopPResetIdsDevice));
    mAllocator->free((void**) (&mSkipDecodeDevice));
    mAllocator->free((void**) (&mSetupWorkspaceDevice));
    std::free(mSkipDecodeHost);
}

template <typename T>
void TopPSamplingLayer<T>::setup(size_t const batchSize, int32_t const* batchSlots, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);

    uint32_t const defaultTopK = 0;
    auto runtimeTopK = setupParams.runtime_top_k.value_or(std::vector<uint32_t>{defaultTopK});
    auto runtimeTopP = setupParams.runtime_top_p.value_or(std::vector<float>{});

    size_t const runtimeTopKSize = runtimeTopK.size();
    size_t const runtimeTopPSize = runtimeTopP.size();

    float const defaultTopPDecay{1.0f};
    auto decayVec = setupParams.top_p_decay.value_or(std::vector<float>(batchSize, defaultTopPDecay));

    float const defaultTopPMin{1e-6f}; // prevent topp becoming 0.0
    auto topPMinVec = setupParams.top_p_min.value_or(std::vector<float>(batchSize, defaultTopPMin));

    int32_t const defaultTopPResetId{-1};
    auto topPResetIdsVec = setupParams.top_p_reset_ids.value_or(std::vector<int32_t>(batchSize, defaultTopPResetId));

    if (runtimeTopPSize == 0)
    {
        for (size_t bi = 0; bi < batchSize; ++bi)
        {
            int32_t bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            mSkipDecodeHost[bid] = true;
        }
        cudaAutoCpy(mSkipDecodeDevice, mSkipDecodeHost, mMaxBatchSize, mStream);
        return;
    }

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }

    for (auto& decay : decayVec)
    {
        if (decay <= 0.f || decay > 1.0f)
        {
            TLLM_LOG_WARNING("Decay (%f) is out of range ([0.0, 1.0f]). Change to 1.0.", decay);
            decay = 1.0f;
        }
    }

    for (auto& topPMin : topPMinVec)
    {
        if (topPMin <= 0.f || topPMin > 1.0f)
        {
            TLLM_LOG_WARNING("TopP min (%f) is out of range ([0.0, 1.0f]). Change to 0.5.", topPMin);
            topPMin = 0.5f;
        }
    }

    uint32_t const topK = runtimeTopK.at(0);
    float const topP = runtimeTopP.at(0);

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopK.size() == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<uint32_t*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<uint32_t*>(mSetupWorkspaceDevice), mRuntimeTopKDevice, batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopP.size() == batchSize,
            fmtstr("runtime_top_p.size() (%lu) == batchSize (%lu) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    auto fillBuffers
        = [this, &batchSize, &batchSlots](std::string name, auto const& vector, auto deviceTmpBuffer, auto deviceBuffer)
    {
        TLLM_CHECK_WITH_INFO(vector.size() == batchSize,
            fmtstr("%s.size() (%lu) == batchSize (%lu) is not satisfied!", name.c_str(), vector.size(), batchSize));
        cudaAutoCpy(deviceTmpBuffer, vector.data(), batchSize, mStream);
        invokeScatterDecodingParams(deviceTmpBuffer, deviceBuffer, batchSlots, batchSize, mStream);
    };

    fillBuffers("top_p_decay", decayVec, reinterpret_cast<float*>(mSetupWorkspaceDevice), mTopPDecayDevice);

    fillBuffers("top_p_min", topPMinVec, reinterpret_cast<float*>(mSetupWorkspaceDevice), mTopPMinDevice);

    fillBuffers(
        "top_p_reset_ids", topPResetIdsVec, reinterpret_cast<int32_t*>(mSetupWorkspaceDevice), mTopPResetIdsDevice);

    {
        dim3 block(std::min((int) batchSize, 256));
        dim3 grid(divUp((int) batchSize, (int) block.x));
        setTopPRuntimeArgs<<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice, runtimeTopKSize, topP,
            mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots, mInitialTopPDevice);
        sync_check_cuda_error();
    }

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mMaxBatchSize, mStream);
    std::vector<float> runtimeTopPs(mMaxBatchSize);
    cudaAutoCpy(runtimeTopPs.data(), mRuntimeTopPDevice, mMaxBatchSize, mStream);
    {
        float maxTopP = 0.f;
        for (size_t bi = 0; bi < batchSize; ++bi)
        {
            int32_t bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            maxTopP = std::max(maxTopP, runtimeTopPs[bid]);
        }
        mRuntimeMaxTopP = std::max(mRuntimeMaxTopP, maxTopP);
    }

    if (!mIsDeterministic)
    {
        int smCnt = mCudaDeviceProp->multiProcessorCount;
        mAirTopPBlockNum = calcAirTopPBlockNum<T, int, float>(batchSize, (int) mVocabSizePadded, smCnt);
    }
}

template <typename T>
void TopPSamplingLayer<T>::forward(DecodingOutputParams& outputs, ForwardParams& inputs)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);

    auto const batchSize = inputs.logits.shape[0];

    // Probabilities must be already computed instead of logits
    auto probs = inputs.logits.template getPtr<T>();
    auto endIds = inputs.end_ids.template getPtr<int const>();
    auto batchSlots = inputs.batch_slots ? inputs.batch_slots->template getPtr<int const>() : nullptr;
    auto curandStatesDevice = inputs.curand_states;
    auto samplingWorkspaceDevice = inputs.sampling_workspace;

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    if (mIsDeterministic)
    {
        invokeTopPInitialize(
            mTopPIdValsDevice, mTopPOffsetDevice, mBeginTopPOffsetDevice, batchSize, mVocabSizePadded, mStream);
        sync_check_cuda_error();
    }

    FinishedState* finishedInput = (inputs.finished)
        ? reinterpret_cast<FinishedState*>(inputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs.finished)
        ? reinterpret_cast<FinishedState*>(outputs.finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;

    float* cumLogProbs = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : nullptr;
    float* outputLogProbs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;
    int* sequenceLength = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    if (mIsDeterministic)
    {
        invokeBatchTopPSampling<T>(samplingWorkspaceDevice, outputs.output_ids_ptr.template getPtr<int*>(),
            sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, probs, mTopPIdValsDevice,
            mTopPOffsetDevice, mBeginTopPOffsetDevice, curandStatesDevice, batchSize, mMaxBatchSize, mVocabSizePadded,
            endIds, mRuntimeMaxTopP, mRuntimeTopPDevice, mStream, mSkipDecodeDevice, batchSlots);
        sync_check_cuda_error();
        invokeComputeToppDecay(mRuntimeTopPDevice, mInitialTopPDevice,
            outputs.output_ids_ptr.template getPtr<int const*>(), mTopPDecayDevice, mTopPMinDevice, mTopPResetIdsDevice,
            sequenceLength, batchSlots, batchSize, mStream);
        sync_check_cuda_error();
    }
    else
    {
        invokeBatchAirTopPSampling<T>(samplingWorkspaceDevice, outputs.output_ids_ptr.template getPtr<int*>(),
            sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, probs, curandStatesDevice,
            batchSize, mMaxBatchSize, mVocabSizePadded, endIds, mRuntimeMaxTopP, mRuntimeTopPDevice, mStream,
            mAirTopPBlockNum, mSkipDecodeDevice, batchSlots);
        sync_check_cuda_error();
    }
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(std::size_t maxBatchSize, std::size_t vocabSize, std::size_t vocabSizePadded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator, hipDeviceProp_t* prop, bool isDeterministic)
    : BaseSamplingLayer<T>(maxBatchSize, vocabSize, vocabSizePadded, stream, std::move(allocator), prop)
    , mIsDeterministic(isDeterministic)
{
    allocateBuffer(mMaxBatchSize);
}

template <typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
