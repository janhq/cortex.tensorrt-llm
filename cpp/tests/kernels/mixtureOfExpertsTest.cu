#include "hip/hip_runtime.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/cutlass_kernels/cutlass_preprocessors.h"
#include "tensorrt_llm/runtime/cudaStream.h"

#include <algorithm>
#include <gtest/gtest.h>
#include <numeric>

#include "tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.h"
#include "tensorrt_llm/runtime/bufferManager.h"

using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

constexpr static float FP8_MAX = 440; // FP8_E4M3_MAX;

template <class T>
__global__ void initWeightsKernel(T* data, int w, int h, float scalar)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
        start_offset[y * w + x] = (x == y) ? T(scalar) : T(0.f);
}

template <class T>
__global__ void initWeightsGatedKernel(T* data, int w, int h, float scalar_1, float scalar_2)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h * 2;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x] = (x == y) ? T(scalar_1) : T(0.f);
        start_offset[(y + h) * w + x] = (x == y) ? T(scalar_2) : T(0.f);
    }
}

template <class T>
__global__ void initBiasToExpertIdKernel(T* data, int w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
        start_offset[x] = T(expert_id);
}

template <class T>
__global__ void initBiasToExpertIdGatedKernel(T* data, int w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w * 2;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
    {
        start_offset[x] = T(expert_id);
        start_offset[x + w] = T(expert_id + 1);
    }
}

#ifdef ENABLE_FP8
using SafeFP8 = __hip_fp8_e4m3_fnuz;
#else
using SafeFP8 = void;
#endif

template <class TypeTuple_>
class MixtureOfExpertsTest : public ::testing::Test
{
protected:
    using DataType = typename TypeTuple_::DataType;
    using WeightType = typename TypeTuple_::WeightType;
    using OutputType = typename TypeTuple_::OutputType;
    constexpr static bool INT4 = std::is_same_v<WeightType, cutlass::uint4b_t>;
    constexpr static bool FP8 = std::is_same_v<DataType, SafeFP8>;
    constexpr static bool INT_QUANT = !std::is_same_v<DataType, WeightType>;
    using WeightStorage = std::conditional_t<INT_QUANT, uint8_t, WeightType>;
    constexpr static int WEIGHT_ELEM_PER_BYTE = INT4 ? 2 : 1;
    int const HIDDEN_SIZE_MULTIPLIER = 1;
    int const DEFAULT_HIDDEN_SIZE = HIDDEN_SIZE_MULTIPLIER * 64 / sizeof(WeightType) * WEIGHT_ELEM_PER_BYTE;

    static BufferManager::CudaStreamPtr mStream;
    static std::unique_ptr<BufferManager> mBufferManager;
    static int mDeviceCount;

    std::vector<BufferManager::IBufferPtr> managed_buffers;
    float* mInputProbabilities{};
    DataType* mInputTensor{};

    int mHiddenSize{};
    int mNumExperts{};
    int mK{};

    float getTolerance(float scale = 1.f)
    {
        float tol = std::is_same_v<DataType, float> ? 0.001
            : std::is_same_v<DataType, half>        ? 0.01
            : std::is_same_v<DataType, SafeFP8>     ? (mIsGated ? 0.06 : 0.02)
                                                    : 0.1;

        // Keep the scale in a sane range
        scale = std::clamp(scale, 1.f, 30.f);
        return scale * tol;
    }

    static bool shouldSkip()
    {
#ifndef ENABLE_FP8
        static_assert(!FP8, "FP8 Tests enabled on unsupported CUDA version");
#endif
        bool should_skip_no_device = mDeviceCount <= 0;
        bool should_skip_unsupported_fp8 = getSMVersion() < 90 && FP8;
        return should_skip_no_device || should_skip_unsupported_fp8;
    }

    static void SetUpTestCase()
    {
        mDeviceCount = getDeviceCount();
        if (shouldSkip())
        {
            GTEST_SKIP();
        }

        mStream = std::make_shared<CudaStream>();
        mBufferManager = std::make_unique<BufferManager>(mStream);
    }

    static void TearDownTestCase()
    {
        mBufferManager.reset();
        mStream.reset();
    }

    void SetUp() override
    {
        assert(mBufferManager);
        if (shouldSkip())
        {
            GTEST_SKIP();
        }
    }

    void TearDown()
    {
        managed_buffers.clear();
    }

    void initWeights(DataType* buffer, int w, int h, float scalar)
    {
        if constexpr (FP8)
            scalar = FP8_MAX; // Automatically set it to max

        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, scalar);
    }

    void initBias(DataType* buffer, int w)
    {
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    void initWeightsGated(DataType* buffer, int w, int h, float scalar_1, float scalar_2)
    {
        if (!mIsGated)
            return initWeights(buffer, w, h, scalar_1);

        if constexpr (FP8)
        {
            float max_scalar = getFP8Scalar(std::max(scalar_1, scalar_2));
            scalar_1 *= max_scalar;
            scalar_2 *= max_scalar;
        }

        h /= 2;
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, scalar_1, scalar_2);
    }

    void initBiasGated(DataType* buffer, int w)
    {
        if (!mIsGated)
            return initBias(buffer, w);

        w /= 2;
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    CutlassMoeFCRunner<DataType, WeightType, OutputType> mMoERunner{};
    char* mWorkspace{};
    float* mScaleProbs{};
    DataType* mRawExpertWeight1{};
    DataType* mRawExpertWeight2{};
    WeightStorage* mExpertWeight1{};
    WeightStorage* mExpertWeight2{};
    DataType* mExpertIntScale1{};
    DataType* mExpertIntScale2{};

    float* mExpertFP8Scale1{};
    float* mExpertFP8Scale2{};
    float* mExpertFP8Scale3{};

    DataType* mExpertBias1{};
    DataType* mExpertBias2{};

    void* mTpExpertScratch{}; // Copy the experts here when slicing up inputs
    size_t mTpExpertScratchSize{};

    OutputType* mFinalOutput{};
    int* mSourceToExpandedMap;
    int* mSelectedExpert;
    bool* mFinished{};
    int mInterSize{};
    int mTotalTokens{};
    int mActiveRows{};

    bool mUseBias = true;

    bool mIsGated = false;
    int mGatedMultiplier = 1;

    tensorrt_llm::ActivationType mActType = tensorrt_llm::ActivationType::Relu;
    MOEExpertScaleNormalizationMode mNormMode = MOEExpertScaleNormalizationMode::NONE;

    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mSelectedConfig = std::nullopt;

    // Keep to simple power of two so we can have tight bounds on precision for quantized modes
    float const mExpertWDiag1{0.5};
    float const mExpertWDiagGated{1};
    float const mExpertWDiag2{2};

    float mMaxInput{};

    template <class T>
    T* allocBuffer(size_t size)
    {
        managed_buffers.emplace_back(mBufferManager->managed(size * sizeof(T)));
        EXPECT_EQ(hipGetLastError(), hipSuccess) << "Error allocating buffer of size: " << size;
        T* ptr = static_cast<T*>(managed_buffers.back()->data());
        return ptr;
    }

    void initBuffersPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int hidden_size, int num_experts, int k,
        std::vector<uint8_t> finished, MOEParallelismConfig parallelism_config)
    {
        managed_buffers.clear();

        mHiddenSize = hidden_size;
        mInterSize = hidden_size * 4;
        mNumExperts = num_experts;
        mK = k;
        mIsGated = tensorrt_llm::isGatedActivation(mActType);
        mGatedMultiplier = mIsGated ? 2 : 1;
        auto const gated_inter = mInterSize * mGatedMultiplier;

        mTotalTokens = 0;
        std::vector<int> h_seq_lens;
        h_seq_lens.push_back(0);
        for (auto& sequence : h_hidden_states)
        {
            assert(sequence.size() % hidden_size == 0);
            int num_tokens = sequence.size() / hidden_size;
            h_seq_lens.emplace_back(h_seq_lens.back() + num_tokens);
            mTotalTokens += num_tokens;
        }

        size_t workspace_size = mMoERunner.getWorkspaceSize(
            mTotalTokens, mHiddenSize, mInterSize, mNumExperts, mK, mActType, parallelism_config);

        auto const stream = mStream->get();

        mWorkspace = allocBuffer<char>(workspace_size);
        // Memset to an obviously incorrect value, so we detect any issues with uninitialised fields
        check_cuda_error(hipMemsetAsync(mWorkspace, 0xD5, workspace_size, stream));
        const size_t expert_matrix_size = mNumExperts * mHiddenSize * mInterSize;

        mRawExpertWeight1 = allocBuffer<DataType>(expert_matrix_size * mGatedMultiplier);
        mRawExpertWeight2 = allocBuffer<DataType>(expert_matrix_size);

        mTpExpertScratchSize = expert_matrix_size * mGatedMultiplier / parallelism_config.tp_size;
        mTpExpertScratchSize += expert_matrix_size / parallelism_config.tp_size;

        mExpertBias1 = nullptr;
        mExpertBias2 = nullptr;
        if (mUseBias)
        {
            // Allow space for the slice of bias1 in the scratch
            mTpExpertScratchSize += mNumExperts * gated_inter / parallelism_config.tp_size;
            mExpertBias1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertBias2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);

            check_cuda_error(hipMemsetAsync(mExpertBias1, 0x0, mNumExperts * gated_inter * sizeof(DataType), stream));
            check_cuda_error(hipMemsetAsync(mExpertBias2, 0x0, mNumExperts * mHiddenSize * sizeof(DataType), stream));
        }

        if constexpr (INT_QUANT)
        {
            mExpertWeight1 = allocBuffer<WeightStorage>(expert_matrix_size * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE);
            mExpertWeight2 = allocBuffer<WeightStorage>(expert_matrix_size / WEIGHT_ELEM_PER_BYTE);

            mTpExpertScratchSize += mNumExperts * gated_inter / parallelism_config.tp_size;
            mExpertIntScale1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertIntScale2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);
        }
        else
        {
            mExpertWeight1 = mRawExpertWeight1;
            mExpertWeight2 = mRawExpertWeight2;
        }

        if constexpr (FP8)
        {
            mExpertFP8Scale1 = allocBuffer<float>(mNumExperts);
            mExpertFP8Scale2 = allocBuffer<float>(1);
            mExpertFP8Scale3 = allocBuffer<float>(mNumExperts);

            ASSERT_NE(mMaxInput, 0.0f);
            initFP8Scales(mMaxInput);
        }

        mTpExpertScratch = nullptr;
        if (parallelism_config.tp_size > 1)
        {
            mTpExpertScratch = allocBuffer<DataType>(mTpExpertScratchSize);
        }

        mActiveRows = mTotalTokens;
        mFinished = nullptr;
        if (!finished.empty())
        {
            mFinished = allocBuffer<bool>(mTotalTokens);
            check_cuda_error(hipMemcpyAsync(
                mFinished, finished.data(), mTotalTokens * sizeof(bool), hipMemcpyHostToDevice, stream));
            static_assert(sizeof(bool) == sizeof(uint8_t), "Test assumes bool is interchangeable with uint8_t");
            mActiveRows = std::count(finished.begin(), finished.end(), 0);
        }

        mInputProbabilities = allocBuffer<float>(mTotalTokens * mNumExperts);
        mScaleProbs = allocBuffer<float>(mTotalTokens * mK);
        mInputTensor = allocBuffer<DataType>(mTotalTokens * mHiddenSize);
        mFinalOutput = allocBuffer<OutputType>(mTotalTokens * mHiddenSize);

        mSourceToExpandedMap = allocBuffer<int>(mTotalTokens * mK);
        mSelectedExpert = allocBuffer<int>(mTotalTokens * mK);

        auto* input_probs_ptr = mInputProbabilities;
        for (auto& sequence : h_router_results)
        {
            check_cuda_error(hipMemcpyAsync(
                input_probs_ptr, sequence.data(), sequence.size() * sizeof(float), hipMemcpyHostToDevice, stream));
            input_probs_ptr += sequence.size();
        }

        auto* hidden_states_ptr = mInputTensor;
        for (auto& sequence : h_hidden_states)
        {
            check_cuda_error(hipMemcpyAsync(hidden_states_ptr, sequence.data(), sequence.size() * sizeof(DataType),
                hipMemcpyHostToDevice, stream));
            hidden_states_ptr += sequence.size();
        }

        check_cuda_error(hipStreamSynchronize(stream));

        // Init the diagonals of our matrix, this will set to the scalar value
        initWeightsGated(mRawExpertWeight1, mHiddenSize, gated_inter, mExpertWDiag1, mExpertWDiagGated);
        initWeights(mRawExpertWeight2, mInterSize, mHiddenSize, mExpertWDiag2);

        if (mUseBias)
        {
            initBiasGated(mExpertBias1, gated_inter);
            initBias(mExpertBias2, mHiddenSize);
        }

        check_cuda_error(hipStreamSynchronize(stream));

        // Runs on the CPU, must be after stream sync
        if constexpr (INT_QUANT)
        {
            cutlass_kernels::QuantType quant_type
                = INT4 ? cutlass_kernels::QuantType::W4_A16 : cutlass_kernels::QuantType::W8_A16;
            std::vector<size_t> shape1 = {(size_t) mNumExperts, (size_t) mHiddenSize, (size_t) gated_inter};
            cutlass_kernels::symmetric_quantize(reinterpret_cast<int8_t*>(mExpertWeight1), mExpertIntScale1,
                mRawExpertWeight1, shape1, quant_type, true);

            std::vector<size_t> shape2 = {(size_t) mNumExperts, (size_t) mInterSize, (size_t) mHiddenSize};
            cutlass_kernels::symmetric_quantize(reinterpret_cast<int8_t*>(mExpertWeight2), mExpertIntScale2,
                mRawExpertWeight2, shape2, quant_type, true);
        }
    }

    constexpr static float getFP8Scalar(float in)
    {
        return FP8_MAX / in;
    }

    void initFP8Scales(float max_input)
    {
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        float maxW1 = mIsGated ? std::max(mExpertWDiag1, mExpertWDiagGated) : mExpertWDiag1;
        float scaleW1 = getFP8Scalar(maxW1);
        float scaleW2 = getFP8Scalar(mExpertWDiag2);
        float scaleAct1 = getFP8Scalar(max_input);

        float maxFC1Output = calcMLPVal(max_input, 0) / mExpertWDiag2;
        float scaleAct2 = getFP8Scalar(maxFC1Output);

        ASSERT_NE(mExpertFP8Scale1, nullptr);
        ASSERT_NE(mExpertFP8Scale2, nullptr);
        ASSERT_NE(mExpertFP8Scale3, nullptr);
        // Dequant values for each expert are 1/(w_i*a_i) calculated above
        std::fill_n(mExpertFP8Scale1, mNumExperts, 1.f / (scaleW1 * scaleAct1));
        std::fill_n(mExpertFP8Scale3, mNumExperts, 1.f / (scaleW2 * scaleAct2));
        *mExpertFP8Scale2 = scaleAct2;

        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    void resetOutBuffers()
    {
        auto stream = mStream->get();

        if (mTpExpertScratch)
            check_cuda_error(hipMemsetAsync(mTpExpertScratch, 0x0, mTpExpertScratchSize, stream));
        check_cuda_error(hipMemsetAsync(mFinalOutput, 0x0, mTotalTokens * mHiddenSize * sizeof(DataType), stream));
        check_cuda_error(hipMemsetAsync(mSourceToExpandedMap, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mSelectedExpert, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mScaleProbs, 0x0, sizeof(float) * mTotalTokens * mK, stream));

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void resizeRouterInputs(std::vector<std::vector<float>>& h_router_results, int num_experts, int num_tokens_per_seq)
    {
        for (int i = 0; i < h_router_results.size(); i++)
        {
            auto& seq_routing = h_router_results[i];
            int num_tokens = num_tokens_per_seq;
            auto hardcoded_experts = seq_routing.size() / num_tokens;
            ASSERT_EQ(seq_routing.size(), hardcoded_experts * num_tokens);
            if (num_experts > hardcoded_experts)
            {
                auto pos = seq_routing.begin() + hardcoded_experts;
                for (int i = 0; i < num_tokens; i++, pos += num_experts)
                {
                    pos = seq_routing.insert(pos, num_experts - hardcoded_experts, 0);
                }
            }
            ASSERT_EQ(seq_routing.size(), num_experts * num_tokens);
        }
    }

    template <class T>
    auto populateTokens(std::vector<T>& hidden_states)
    {
        if constexpr (std::is_same_v<T, SafeFP8>)
        {
            std::vector<OutputType> internal_states(hidden_states.size());
            populateTokens(internal_states);

            mMaxInput = *std::max_element(internal_states.begin(), internal_states.end());
            float scalar = getFP8Scalar(mMaxInput);
            std::transform(internal_states.begin(), internal_states.end(), hidden_states.begin(),
                [scalar](OutputType in) -> T { return static_cast<T>((float) in * scalar); });
            // Do the reverse transformation since we only have so much precision and this is a pretty broad range
            std::transform(hidden_states.begin(), hidden_states.end(), internal_states.begin(),
                [scalar](T in) -> OutputType { return static_cast<OutputType>(((float) in) / scalar); });
            return internal_states;
        }
        else
        {
            std::iota(hidden_states.begin(), hidden_states.end(), 0.0f);
            // Lambda subtracts a small value so we have some < 0 to test the activation for negatives
            std::transform(hidden_states.begin(), hidden_states.end(), hidden_states.begin(),
                [l = hidden_states.size()](auto a) { return a / (T) l - T(0.01f); });
            return hidden_states;
        }
    }

    void runMoEPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int hidden_size, int num_experts, int k,
        std::vector<uint8_t> finished = {}, MOEParallelismConfig parallelism_config = {})
    {
        initBuffersPermute(std::move(h_hidden_states), std::move(h_router_results), hidden_size, num_experts, k,
            finished, parallelism_config);
        runMoEPermute(parallelism_config);
    }

    auto getWeights(MOEParallelismConfig parallelism_config)
    {
        void* scale_1 = FP8 ? (void*) mExpertFP8Scale1 : (void*) mExpertIntScale1;
        void* scale_2 = FP8 ? (void*) mExpertFP8Scale2 : (void*) mExpertIntScale2;
        void* scale_3 = FP8 ? mExpertFP8Scale3 : nullptr;

        if (parallelism_config.tp_size > 1)
        {
            int const tp_size = parallelism_config.tp_size;
            int const tp_rank = parallelism_config.tp_rank;

            const size_t matrix_size = mHiddenSize * mInterSize / tp_size;
            const size_t gated_matrix_size = mHiddenSize * mInterSize * mGatedMultiplier / tp_size;
            const size_t row_size_inter = mInterSize / tp_size;
            const size_t gated_row_size_inter = mInterSize * mGatedMultiplier / tp_size;
            const size_t gated_bias_size = mUseBias ? gated_row_size_inter : 0;

            auto* weight_1 = reinterpret_cast<WeightStorage*>(mTpExpertScratch);
            auto* weight_2 = weight_1 + mNumExperts * gated_matrix_size;
            auto* bias_1 = reinterpret_cast<DataType*>(weight_2 + mNumExperts * matrix_size);
            auto* int_scale_1 = bias_1 + mNumExperts * gated_bias_size;

            // 2D memcpy just the slices we care about
            // TODO Re-quantize here with matrices divided
            const size_t row_size_1 = matrix_size * sizeof(WeightStorage) / WEIGHT_ELEM_PER_BYTE;
            check_cuda_error(hipMemcpy2DAsync(weight_1, row_size_1, (uint8_t*) mExpertWeight1 + row_size_1 * tp_rank,
                row_size_1 * tp_size, row_size_1, mNumExperts * mGatedMultiplier, hipMemcpyDeviceToDevice,
                mStream->get()));

            const size_t row_size_2 = row_size_inter * sizeof(WeightStorage) / WEIGHT_ELEM_PER_BYTE;
            check_cuda_error(hipMemcpy2DAsync(weight_2, row_size_2, (uint8_t*) mExpertWeight2 + row_size_2 * tp_rank,
                row_size_2 * tp_size, row_size_2, mNumExperts * mHiddenSize, hipMemcpyDeviceToDevice, mStream->get()));

            if (mUseBias)
            {
                const size_t row_size_bias = row_size_inter * sizeof(DataType);
                check_cuda_error(hipMemcpy2DAsync(bias_1, row_size_bias,
                    (uint8_t*) mExpertBias1 + row_size_bias * tp_rank, row_size_bias * tp_size, row_size_bias,
                    mNumExperts * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));
            }

            if constexpr (INT_QUANT)
            {
                scale_2 = mExpertIntScale2;
                const size_t row_size_scale = row_size_inter * sizeof(DataType);
                check_cuda_error(hipMemcpy2DAsync(scale_1, row_size_scale,
                    (uint8_t*) mExpertIntScale1 + row_size_scale * tp_rank, row_size_scale * tp_size, row_size_scale,
                    mNumExperts * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));
            }

            bias_1 = mUseBias ? bias_1 : nullptr;
            return std::tuple{weight_1, weight_2, bias_1, mExpertBias2, scale_1, scale_2, scale_3};
        }
        else if (parallelism_config.ep_size > 1)
        {
            const size_t gated_inter = mInterSize * mGatedMultiplier;
            const size_t experts_per_node = mNumExperts / parallelism_config.ep_size;
            const size_t weight_matrix_size = mHiddenSize * mInterSize * experts_per_node / WEIGHT_ELEM_PER_BYTE;
            const size_t bias_fc1_size = gated_inter * experts_per_node;
            const size_t bias_fc2_size = mHiddenSize * experts_per_node;
            const size_t scale1_size = gated_inter * experts_per_node;
            const size_t scale2_size = mHiddenSize * experts_per_node;
            auto* weight1_ptr = mExpertWeight1 + weight_matrix_size * mGatedMultiplier * parallelism_config.ep_rank;
            auto* weight2_ptr = mExpertWeight2 + weight_matrix_size * parallelism_config.ep_rank;
            auto* bias1_ptr = mUseBias ? mExpertBias1 + bias_fc1_size * parallelism_config.ep_rank : nullptr;
            auto* bias2_ptr = mUseBias ? mExpertBias2 + bias_fc2_size * parallelism_config.ep_rank : nullptr;

            if (INT_QUANT)
            {
                scale_1 = mExpertIntScale1 + scale1_size * parallelism_config.ep_rank;
                scale_2 = mExpertIntScale2 + scale2_size * parallelism_config.ep_rank;
            }
            if constexpr (FP8)
            {
                scale_1 = mExpertFP8Scale1 + experts_per_node * parallelism_config.ep_rank;
                scale_3 = mExpertFP8Scale3 + experts_per_node * parallelism_config.ep_rank;
            }

            return std::tuple{weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr, scale_1, scale_2, scale_3};
        }

        return std::tuple{mExpertWeight1, mExpertWeight2, mExpertBias1, mExpertBias2, scale_1, scale_2, scale_3};
    }

    void runMoEPermute(MOEParallelismConfig parallelism_config)
    {
        // Clear the buffers to blank so we can assume zero if not written
        resetOutBuffers();

        auto const [weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr, scale1_ptr, scale2_ptr, scale3_ptr]
            = getWeights(parallelism_config);

        auto stream = mStream->get();
        auto tactic = mSelectedConfig;
        if (!tactic)
        {
            tactic = mMoERunner.getTactics()[0];
        }

        QuantParams quant_params;
        if constexpr (INT_QUANT)
        {
            quant_params = QuantParams::Int(scale1_ptr, scale2_ptr);
        }
        else
        {
            quant_params = QuantParams::FP8(static_cast<float const*>(scale1_ptr),
                static_cast<float const*>(scale2_ptr), static_cast<float const*>(scale3_ptr));
        }

        mMoERunner.setTactic(tactic);
        mMoERunner.runMoe(mInputTensor, mInputProbabilities, weight1_ptr, bias1_ptr, mActType, weight2_ptr, bias2_ptr,
            quant_params, mTotalTokens, mHiddenSize, mInterSize / parallelism_config.tp_size, mNumExperts, mK,
            mWorkspace, mFinalOutput, mFinished, mActiveRows, mScaleProbs, mSourceToExpandedMap, mSelectedExpert,
            parallelism_config, mNormMode, stream);

        check_cuda_error(hipStreamSynchronize(stream));
    }

    template <class T>
    std::vector<T> getDataFromDevice(T const* in, size_t length)
    {
        std::vector<T> data(length);

        auto const stream = mStream->get();
        check_cuda_error(hipMemcpyAsync(data.data(), in, length * sizeof(T), hipMemcpyDeviceToHost, stream));
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        return data;
    }

    auto maskSelectedExpertsForTP(std::vector<int> const& vector, int tp_size, int tp_rank)
    {
        std::vector<int> result;
        int num_experts_per_node = mNumExperts / tp_size;
        std::transform(vector.begin(), vector.end(), std::back_inserter(result),
            [=](int entry)
            {
                if (entry >= num_experts_per_node * tp_rank && entry < num_experts_per_node * (tp_rank + 1))
                    return entry;
                return mNumExperts;
            });
        return result;
    }

    void debugPrint()
    {
#define PRINT_CAST(array, size, cast)                                                                                  \
    do                                                                                                                 \
        if (array)                                                                                                     \
        {                                                                                                              \
            auto data = getDataFromDevice(array, size);                                                                \
            std::cout << #array << ": ";                                                                               \
            for (auto v : data)                                                                                        \
            {                                                                                                          \
                if (cast(v))                                                                                           \
                    std::cout << cast(v) << ", ";                                                                      \
                else                                                                                                   \
                    std::cout << "., ";                                                                                \
            }                                                                                                          \
            std::cout << std::endl;                                                                                    \
        }                                                                                                              \
    while (0)
#define PRINT(array, size) PRINT_CAST(array, size, )

        using WeightPrintType = std::conditional_t<INT_QUANT, uint8_t, WeightStorage>;
        PRINT_CAST((WeightPrintType*) mExpertWeight1,
            mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE, float);
        PRINT_CAST(
            (WeightPrintType*) mExpertWeight2, mNumExperts * mHiddenSize * mInterSize / WEIGHT_ELEM_PER_BYTE, float);
        // PRINT_CAST(mRawExpertWeight1, mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier, float);
        // PRINT_CAST(mRawExpertWeight2, mNumExperts * mHiddenSize * mInterSize, float);
        PRINT_CAST(mExpertBias1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertBias2, mNumExperts * mHiddenSize, float);
        PRINT_CAST(mExpertIntScale1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertIntScale2, mNumExperts * mHiddenSize, float);
        PRINT(mFinalOutput, mTotalTokens * mHiddenSize);
        PRINT_CAST((uint8_t*) mFinished, mTotalTokens, (int) );
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT(mScaleProbs, mTotalTokens * mK);
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT_CAST(mInputTensor, mTotalTokens * mHiddenSize, float);
        PRINT(mSourceToExpandedMap, mTotalTokens * mK);
        PRINT(mSelectedExpert, mTotalTokens * mK);

#undef PRINT_CAST
#undef PRINT
    }

    template <class T>
    T actfn(T in)
    {
        if (mActType == tensorrt_llm::ActivationType::Identity)
            return in;
        if (mActType == tensorrt_llm::ActivationType::Relu)
            return std::max(in, T(0.0f));
        if (mActType == tensorrt_llm::ActivationType::Gelu || mActType == tensorrt_llm::ActivationType::Geglu)
            return (std::erf(float(in) * float(sqrt(0.5))) + 1) * 0.5f * float(in);
        assert(false);
        return in;
    }

    float calcMLPVal(float input, int expert_id, bool final_bias = false)
    {
        if (expert_id >= mNumExperts)
            return 0;

        float w1_bias = mUseBias ? expert_id : 0.f;
        float activated = 0;
        if (mIsGated)
        {
            float scalar = mExpertWDiag1;
            float fc1 = (float) input * scalar + (float) w1_bias;

            float gated_scalar = mExpertWDiagGated;
            float gated_bias = mUseBias ? (float) w1_bias + 1.f : 0.f;
            float gate = (float) input * gated_scalar + gated_bias;

            activated = fc1 * actfn(gate);
        }
        else
        {
            float fc1 = input * mExpertWDiag1 + w1_bias;
            activated = actfn(fc1);
        }

        EXPECT_TRUE(mUseBias || !final_bias);
        return activated * mExpertWDiag2 + (float) (final_bias ? expert_id : 0);
    }

    float calcMLPValWithFinalBias(float input, int expert_id)
    {
        return calcMLPVal(input, expert_id, mUseBias);
    }

    // NOTE This is a useful function for debugging routing failures. But you need to know the exact offset of
    //   this info in the workspace so having a test depend on something so internal is suboptimal
    //
    // void comparePermuted(const std::vector<int>& expected_experts, const std::vector<int>& expected_permutation,
    //     const std::vector<DataType>& input_data)
    //{
    //     auto states = getDataFromDevice(magic incantation into workspace, mTotalTokens * mK * mHiddenSize);
    //
    //    // Loop for the number of times each token is duplicated
    //    for (int k_idx = 0; k_idx < mK; k_idx++)
    //    {
    //        for (int token_id = 0; token_id < mTotalTokens; token_id++)
    //        {
    //            // Permutation has the position of the first copy of all token,
    //            // followed by the position of the second copy of all tokens etc.
    //            const int permuted_position = expected_permutation[k_idx * mTotalTokens + token_id];
    //
    //            // Expected experts has all the selected experts for token one,
    //            // followed by all the selected experts for token two etc.
    //            const int expert_id = expected_experts[token_id * mK + k_idx];
    //
    //            // Compare the copied tokens with the projection applied
    //            for (int hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
    //            {
    //                auto ref = calcMLPVal(input_data[token_id * mHiddenSize + hidden_id], expert_id);
    //                auto actual = states[permuted_position * mHiddenSize + hidden_id];
    //                ASSERT_NEAR(ref, actual, getTolerance(ref))
    //                    << "Incorrect value at position: mK: " << k_idx << ", token: " << token_id
    //                    << ", permuted dest: " << permuted_position << ", expert id: " << expert_id
    //                    << ", hidden id: " << hidden_id;
    //            }
    //        }
    //    }
    //}

    std::vector<float> softmax(std::vector<float> const& expected_probs)
    {
        std::vector<float> softmax;
        // All values we test are 0-1 so we can skip the normalization step
        std::transform(expected_probs.begin(), expected_probs.end(), std::back_inserter(softmax),
            [&](float const in) -> float
            {
                auto res = exp(in);
                return res;
            });

        for (int token = 0; token < mTotalTokens; token++)
        {
            auto start = softmax.begin() + token * mNumExperts;
            auto end = start + mNumExperts;
            auto sum = std::accumulate(start, end, 0.f);
            std::transform(start, end, start, [=](auto in) { return in / sum; });
        }

        return softmax;
    }

    void compareSoftmax(std::vector<int> const& expected_experts, std::vector<float> const& expected_probs,
        std::vector<float> scale_probs = {})
    {
        if (scale_probs.empty())
            scale_probs = getDataFromDevice(mScaleProbs, mTotalTokens * mK);
        auto softmax_probs = softmax(expected_probs);
        for (int token_id = 0; token_id < mTotalTokens; token_id++)
        {
            for (int k_idx = 0; k_idx < mK; k_idx++)
            {
                int selected_expert = expected_experts[token_id * mK + k_idx];
                if (selected_expert < mNumExperts) // Ignore 'finished' values
                {
                    ASSERT_NEAR(softmax_probs[token_id * mNumExperts + selected_expert],
                        scale_probs[token_id * mK + k_idx], getTolerance())
                        << "Scales mismatched for token: " << token_id << " k: " << k_idx
                        << " selected_expert: " << selected_expert;
                }
            }
        }
    }

    void renormScales(float* probs, int const* experts)
    {
        if (mNormMode == MOEExpertScaleNormalizationMode::NONE)
            return;
        float sum = 0;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            sum += probs[experts[k_idx]];
        }
        float norm_factor = 1.0f / sum;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            probs[experts[k_idx]] *= norm_factor;
        }
    }

    void compareFinal(std::vector<int> const& expected_experts, std::vector<float> const& expected_probs,
        std::vector<OutputType> const& input_data, std::vector<OutputType> final_results = {})
    {
        if (final_results.empty())
            final_results = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);

        auto softmax_probs = softmax(expected_probs);
        for (int token_id = 0; token_id < mTotalTokens; token_id++)
        {
            renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

            for (int hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
            {
                float sum = 0.0f;
                // Loop for the number of times each token is duplicated
                for (int k_idx = 0; k_idx < mK; k_idx++)
                {
                    int selected_expert = expected_experts[token_id * mK + k_idx];
                    sum += float(calcMLPValWithFinalBias(
                               static_cast<float>(input_data[token_id * mHiddenSize + hidden_id]), selected_expert))
                        * softmax_probs[token_id * mNumExperts + selected_expert];
                }

                ASSERT_NEAR(OutputType{sum}, final_results[token_id * mHiddenSize + hidden_id], getTolerance(sum))
                    << "Incorrect final value at position: " << token_id * mHiddenSize + hidden_id;
            }
        }
    }

    void BasicPermuteTest(int k = 1);

    std::vector<int> calcPermuteMapExpertParallel(std::vector<int> const& expected_experts);
    void ExpertParallelTest(int k = 1);

    void TensorParallelTest(int k = 1);
};

template <class DataType_, class WeightType_ = DataType_, class OutputType_ = DataType_>
struct WeightParams
{
    using DataType = DataType_;
    using WeightType = WeightType_;
    using OutputType = OutputType_;
};

// TODO Fix int quantized
using Types = ::testing::Types<
#ifdef ENABLE_BF16
    WeightParams<__hip_bfloat16>,
#endif
#ifdef ENABLE_FP8
    WeightParams<SafeFP8, SafeFP8, half>,
#endif
    WeightParams<half>, WeightParams<float>

    //, WeightParams<half, uint8_t>, WeightParams<half, cutlass::uint4b_t>

    >;
TYPED_TEST_SUITE(MixtureOfExpertsTest, Types);

template <class TypeParam_>
BufferManager::CudaStreamPtr MixtureOfExpertsTest<TypeParam_>::mStream{};
template <class TypeParam_>
std::unique_ptr<BufferManager> MixtureOfExpertsTest<TypeParam_>::mBufferManager{};
template <class TypeParam_>
int MixtureOfExpertsTest<TypeParam_>::mDeviceCount{};

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::BasicPermuteTest(int k)
{
    if constexpr (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int num_experts = 4;
    int num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<std::vector<DataType>> hidden_input = {hidden_states};
    std::vector<std::vector<float>> router_input = {probs};
    resizeRouterInputs(router_input, num_experts, num_tokens);

    runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k);

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};

    auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
    EXPECT_EQ(selected_expert, expected_experts);

    auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
    // This is the final position of:
    // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E2
    std::vector<int> permute_map{0, 2, 1};
    if (k == 2)
        permute_map = {0, 5, 4, 3, 2, 1};
    if (k == 3)
        permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};
    ASSERT_EQ(permute_map, proj_map);
    compareSoftmax(selected_expert, router_input[0]);
    compareFinal(selected_expert, router_input[0], raw_unquant_input);
}

TYPED_TEST(MixtureOfExpertsTest, Permute)
{
    this->BasicPermuteTest();
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK2)
{
    this->BasicPermuteTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK3)
{
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNoBias)
{
    this->mUseBias = false;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteRenormalization)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, Finished)
{
    if (this->FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        this->mUseBias = false;
    }

    using DataType = typename TypeParam::DataType;
    int hidden_size = this->DEFAULT_HIDDEN_SIZE;
    int num_experts = 4;
    int num_tokens = 3;
    int k = 2;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = this->populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15, //
        0.05, 0.2, 0.05, 0.7, //
        0.25, 0.2, 0.35, 0.2, //
    };

    this->runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {0, 0, 1});

    auto selected_expert = this->getDataFromDevice(this->mSelectedExpert, num_tokens * k);
    // Token 1
    EXPECT_EQ(selected_expert[0], 0);
    EXPECT_EQ(selected_expert[1], 2);
    // Token 2
    EXPECT_EQ(selected_expert[2], 3);
    EXPECT_EQ(selected_expert[3], 1);
    // Token 3
    EXPECT_EQ(selected_expert[4], num_experts); // One past the end
    EXPECT_EQ(selected_expert[5], num_experts);

    auto proj_map = this->getDataFromDevice(this->mSourceToExpandedMap, num_tokens * k);
    // This is the final position of:
    // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E3
    std::vector<int> permute_map{0, 3, 4, 2, 1, 5};
    ASSERT_EQ(permute_map, proj_map);
    this->compareSoftmax(selected_expert, probs);
    this->compareFinal(selected_expert, probs, raw_unquant_input);
}

template <class TypeParam_>
std::vector<int> MixtureOfExpertsTest<TypeParam_>::calcPermuteMapExpertParallel(
    std::vector<int> const& expected_experts)
{
    std::vector<int> map(expected_experts.size());
    auto getInterleavedIndex = [this](int i) { return (i % mK) * mTotalTokens + i / mK; };
    int map_idx = 0;
    for (int expert = 0; expert <= mNumExperts; expert++)
    {
        for (int i = 0; i < map.size(); i++)
        {
            if (expected_experts[i] == expert)
                map[getInterleavedIndex(i)] = map_idx++;
        }
    }

    return map;
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::ExpertParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int parallelism = 2;
    int num_experts = 4;
    int num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
    std::vector<OutputType> results(hidden_states.size(), 0);
    for (int i = 0; i < parallelism; i++)
    {
        if (i == 0)
        {
            // Only need to init the inputs on the first iteration
            runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                MOEParallelismConfig::ExpertParallelism(parallelism, i));
        }
        else
        {
            runMoEPermute(MOEParallelismConfig::ExpertParallelism(parallelism, i));
        }

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        // Experts should only be selected when we are on the right node
        // Note the index is [0,num_experts_per_node), so we offset the experts by the start for this node
        int const start_expert = i * (mNumExperts / parallelism);
        std::transform(selected_expert.begin(), selected_expert.end(), selected_expert.begin(),
            [&](int val) { return val == mNumExperts ? mNumExperts : val + start_expert; });
        auto masked_expected_experts = maskSelectedExpertsForTP(expected_experts, parallelism, i);
        ASSERT_EQ(selected_expert, masked_expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        auto permute_map = calcPermuteMapExpertParallel(masked_expected_experts);
        ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;
        compareSoftmax(expected_experts, probs);

        // Do the final reduce
        auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
        std::transform(iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
    }

    compareFinal(expected_experts, probs, raw_unquant_input, results);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallel)
{
    this->ExpertParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelK2)
{
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelNoBias)
{
    this->mUseBias = false;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::TensorParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    int hidden_size = DEFAULT_HIDDEN_SIZE;
    int parallelism = 8;
    int num_experts = 4;
    int num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
    std::vector<OutputType> results(hidden_states.size(), 0);
    for (int i = 0; i < parallelism; i++)
    {
        if (i == 0)
        {
            // Only need to init the inputs on the first iteration
            runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                MOEParallelismConfig::TensorParallelism(parallelism, i));
        }
        else
        {
            runMoEPermute(MOEParallelismConfig::TensorParallelism(parallelism, i));
        }

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        EXPECT_EQ(selected_expert, expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        std::vector<int> permute_map{0, 2, 1};
        if (k == 2)
            permute_map = {0, 5, 4, 3, 2, 1};
        if (k == 3)
            permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};

        ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;

        // Do the final reduce
        auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
        std::transform(iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
    }

    compareFinal(expected_experts, probs, raw_unquant_input, results);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallel)
{
    this->TensorParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelK2)
{
    this->TensorParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelK3)
{
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelNoBias)
{
    this->mUseBias = false;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, ConfigSweep)
{
    auto configs = this->mMoERunner.getTactics();
    for (auto conf : configs)
    {
        using namespace tensorrt_llm::cutlass_extensions;
        std::stringstream tactic;
        tactic << "Failed " << (conf.is_sm90 ? "SM90+" : "<SM90") << " tactic with tile shape ";
        if (conf.tile_config_sm90 != CutlassTileConfigSM90::ChooseWithHeuristic)
        {
            tactic << (int) conf.tile_config_sm90 << " and cluster shape " << (int) conf.cluster_shape
                   << " mainloop sched " << (int) conf.mainloop_schedule << " epi sched "
                   << (int) conf.epilogue_schedule;
        }
        else if (conf.tile_config != CutlassTileConfig::ChooseWithHeuristic)
        {
            tactic << (int) conf.tile_config << " and stages " << (int) conf.stages << " split k "
                   << (int) conf.split_k_factor;
        }
        else
        {
            FAIL() << "Uninitialised tactic encountered";
        }

        EXPECT_NO_THROW({
            this->mSelectedConfig = conf;
            this->BasicPermuteTest();
            if (::testing::Test::HasFailure())
                throw std::runtime_error("Test Failed");
        }) << tactic.str();
    }
}
